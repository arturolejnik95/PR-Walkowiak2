#include "hip/hip_runtime.h"
#define GRID_HEIGHT (MATRIX_SIZE / BLOCK_SIZE)
#define GRID_WIDTH (GRID_HEIGHT)


__global__ void matrixMulCUDA(float *C, float *A, float *B) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = MATRIX_SIZE * BLOCK_SIZE * by;
    int aEnd = aBegin + MATRIX_SIZE - 1;
    int aStep = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep = BLOCK_SIZE * MATRIX_SIZE;

    float C_local = 0;

    // macierze na których wykonujemy obliczenia
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // macierze do których równolegle z obliczeniami wpisujemy dane kolejnych bloków
    __shared__ float A_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shared[BLOCK_SIZE][BLOCK_SIZE];

    A_shared[ty][tx] = A[aBegin + MATRIX_SIZE * ty + tx];
    B_shared[ty][tx] = B[bBegin + MATRIX_SIZE * ty + tx];

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        //Pobieranie danych
        As[ty][tx] = A_shared[ty][tx];
        Bs[ty][tx] = B_shared[ty][tx];


        __syncthreads();

        A_shared[ty][tx] = A[a + MATRIX_SIZE * ty + tx];
        B_shared[ty][tx] = B[b + MATRIX_SIZE * ty + tx];

#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            //Kolumna razy wiersz
            C_local += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
        
    }

    int c = MATRIX_SIZE * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + MATRIX_SIZE * ty + tx] = C_local;
}

// wywołanie
dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
dim3 grid(GRID_WIDTH, GRID_HEIGHT);
matrixMulCUDA<<< grid, threads >>>(d_C, d_A, d_B);